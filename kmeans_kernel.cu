
#include <hip/hip_runtime.h>
#include <iostream>
#include <cfloat>
#include <cstdio>
#include <vector>
#include <cmath>

// returns the double distance between 2 dims-dimensional points
__device__ double calcDistanceCuda(double* p1, double* p2, int dims) {
    double sum = 0.0;
    for(int i=0;i<dims;i++) {
        sum += pow(p1[i]-p2[i], 2);
    }
    return sqrt(sum);
}

// returns index of minimum distance in the array (will be the cluster number)
__device__ int findMinDistanceCuda(double* distances, int num_cluster) {
    double minDist = DBL_MAX;
    int index = -1;
    for(int i=0;i<num_cluster;i++) {
        double distance = distances[i];
        if(distance<minDist) {
            minDist = distance;
            index = i;
        }
    }
    return index;
}

// d_distances: num_points x num_cluster [flattened]
// will be called with <<<num_points, num_cluster>>>
__global__ void calcDistances(double* d_distances, double* d_points, double* d_centroids, int dims, int num_cluster) {
    d_distances[blockIdx.x*num_cluster+threadIdx.x] = calcDistanceCuda(&d_points[blockIdx.x*dims], &d_centroids[threadIdx.x*dims], dims);
}

// take argmax based on index
// iterates over num_points values
__global__ void updateLabels(int* d_labels, double* d_distances, int num_cluster, int num_points) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index<num_points) {
        d_labels[index] = findMinDistanceCuda(&d_distances[index*num_cluster], num_cluster);
    }
}

void findNearestCentroidsCuda(double* d_points, int* d_labels, double* d_centroids, int num_points, int num_cluster, int dims) {
    // malloc distances array
    double* d_distances;
    hipMalloc((void**)&d_distances, num_points * num_cluster * sizeof(double));

    // calc all distances (writing to d_distances)
    calcDistances<<<num_points, num_cluster>>>(d_distances, d_points, d_centroids, dims, num_cluster);    
    hipDeviceSynchronize();

    // take argmax based on index (reading from d_distances)
    updateLabels<<<(num_points+32-1)/32, 32>>>(d_labels, d_distances, num_cluster, num_points);
    hipDeviceSynchronize();

    // free distances memory
    hipFree(d_distances);
}

// found this implementation online, this is not my function!
__device__ double doubleAtomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

// will be called with <<<num_points, dims>>>
__global__ void sumPointsAcrossLabels(int* d_labels, double* d_centroids, double* d_points, int dims) {
    int row = blockIdx.x;
    int col = threadIdx.x;
    int label = d_labels[row];
    doubleAtomicAdd(&d_centroids[label*dims+col], d_points[row*dims+col]);
}

// NOTE: will be called with <<<1, num_points>>>
// have num_points things going
__global__ void sumLabelFreqs(int* d_labels, int* d_freqs, int num_points) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < num_points) {
        int label = d_labels[index];
        atomicAdd(&d_freqs[label], 1);
    }
}

// will be called with <<<1, num_points>>> [mathy]
// shared memory section: num_points*sizeof(int)
__global__ void sumLabelFreqsShmem(int* d_labels, int* d_freqs, int num_points) {
    extern __shared__ int s_labels[];
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int lid = threadIdx.x;
    if(index < num_points) {
        // copy over data to shared memory
        s_labels[lid] = d_labels[index];
        __syncthreads();

        int label = s_labels[lid];
        atomicAdd(&d_freqs[label], 1);
    }
}

// will be called with <<<num_cluster, dims>>>
__global__ void divideAcrossLabels(double* d_centroids, int* d_freqs, int dims) {
    int row = blockIdx.x;
    int col = threadIdx.x;
    d_centroids[row*dims+col] /= d_freqs[row];
}

void averageLabeledCentroidsCuda(double* d_points, int* d_labels, int num_cluster, int num_points, double* d_centroids, int dims, bool shared) {
    // zero out all the centroid values
    hipMemset(d_centroids, 0.0, num_cluster * dims * sizeof(double));
    hipDeviceSynchronize();
    
    // sum up across labels, track frequencies
    int *d_freqs;
    hipMalloc((void**)&d_freqs, num_cluster * sizeof(int));
    hipMemset(d_freqs, 0, num_cluster * sizeof(int));
    hipDeviceSynchronize();
    
    sumPointsAcrossLabels<<<num_points, dims>>>(d_labels, d_centroids, d_points, dims);
    if(shared) {
        sumLabelFreqsShmem<<<(num_points+32-1)/32, 32, num_points*sizeof(int)>>>(d_labels, d_freqs, num_points);
    } else {
        sumLabelFreqs<<<(num_points+32-1)/32, 32>>>(d_labels, d_freqs, num_points);
    }
    
    hipDeviceSynchronize();

    // divide each centroid value by its frequency
    divideAcrossLabels<<<num_cluster, dims>>>(d_centroids, d_freqs, dims);
    hipDeviceSynchronize();
    hipFree(d_freqs);
}

// will be called with <<<1, num_cluster>>>
// arr[0] will contain the number of distances above the threshold
__global__ void checkDistanceAboveThresh(int* arr, double threshold, int dims, int num_cluster, double* d_centroids, double* d_oldCentroids) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < num_cluster) {
        if(calcDistanceCuda(&d_centroids[index*dims], &d_oldCentroids[index*dims], dims)>threshold) atomicAdd(&arr[0], 1);
    }
}

bool convergedCuda(double* d_centroids, double* d_oldCentroids, double threshold, int num_cluster, int dims) {
    // memory needed
    int *d_arr;
    hipMalloc((void**)&d_arr, sizeof(int));
    int arr;

    // check distance, increment d_arr
    checkDistanceAboveThresh<<<1, num_cluster>>>(d_arr, threshold, dims, num_cluster, d_centroids, d_oldCentroids);
    hipDeviceSynchronize();

    // copy back d_arr value to host 
    hipMemcpy(&arr, d_arr, sizeof(int), hipMemcpyDeviceToHost);
    bool hasConverged = arr == 0;

    // cleanup + return
    hipFree(d_arr);
    return hasConverged;
}

// returns float*, [0]: mem_overhead_time, [1]: algo_time
float* gpu_kmeans(double** centroids, double** old_centroids, double** points, int* labels, double threshold, int num_cluster, int dims, int max_num_iter, int num_points, bool shared) {
    // time measurement
    float temp = 0;
    hipEvent_t mem_overhead_start, mem_overhead_stop;
    hipEventCreate(&mem_overhead_start);
    hipEventCreate(&mem_overhead_stop);
    float mem_overhead_time = 0;
    hipEvent_t algo_start, algo_stop;
    hipEventCreate(&algo_start);
    hipEventCreate(&algo_stop);
    float algo_time = 0;

    // allocate device memory & copy over data
    hipEventRecord(mem_overhead_start);
    double *d_points, *d_centroids, *d_old_centroids;
    int *d_labels;

    hipMalloc((void**)&d_points, num_points * dims * sizeof(double));
    for(int i = 0; i < num_points; i++) {
        hipMemcpy(&d_points[i*dims], points[i], dims * sizeof(double), hipMemcpyHostToDevice);
    }
    hipMalloc((void**)&d_centroids, num_cluster * dims * sizeof(double));
    for(int i = 0; i < num_cluster; i++) {
        hipMemcpy(&d_centroids[i*dims], centroids[i], dims * sizeof(double), hipMemcpyHostToDevice);
    }
    hipMalloc((void**)&d_old_centroids, num_cluster * dims * sizeof(double));
    for(int i = 0; i < num_cluster; i++) {
        hipMemcpy(&d_old_centroids[i*dims], old_centroids[i], dims * sizeof(double), hipMemcpyHostToDevice);
    }
    hipMalloc((void**)&d_labels, num_points * sizeof(int));
    hipMemcpy(d_labels, labels, num_points * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(mem_overhead_stop);
    hipEventSynchronize(mem_overhead_stop);
    hipEventElapsedTime(&temp, mem_overhead_start, mem_overhead_stop);
    mem_overhead_time += temp;

    // run loop
    hipEventRecord(algo_start);
    int iteration = 0;
    bool done = iteration >= max_num_iter || convergedCuda(d_centroids, d_old_centroids, threshold, num_cluster, dims);
    while(!done) { 
        hipMemcpy(d_old_centroids, d_centroids, num_cluster * dims * sizeof(double), hipMemcpyDeviceToDevice);
        iteration++;
        findNearestCentroidsCuda(d_points, d_labels, d_centroids, num_points, num_cluster, dims);
        averageLabeledCentroidsCuda(d_points, d_labels, num_cluster, num_points, d_centroids, dims, shared);
        done = iteration >= max_num_iter || convergedCuda(d_centroids, d_old_centroids, threshold, num_cluster, dims);
    }
    hipEventRecord(algo_stop);
    hipEventSynchronize(algo_stop);
    hipEventElapsedTime(&temp, algo_start, algo_stop);
    algo_time = temp;

    hipEventRecord(mem_overhead_start);
    // copy over data back to host memory
    for(int i = 0; i < num_points; i++) {
        hipMemcpy(points[i], &d_points[i*dims], dims * sizeof(double), hipMemcpyDeviceToHost);
    }
    for(int i = 0; i < num_cluster; i++) {
        hipMemcpy(centroids[i], &d_centroids[i*dims], dims * sizeof(double), hipMemcpyDeviceToHost);
    }
    for(int i = 0; i < num_cluster; i++) {
        hipMemcpy(old_centroids[i], &d_old_centroids[i*dims], dims * sizeof(double), hipMemcpyDeviceToHost);
    }
    hipMemcpy(labels, d_labels, num_points * sizeof(int), hipMemcpyDeviceToHost);

    // free all device memory
    hipFree(d_points);
    hipFree(d_centroids);
    hipFree(d_old_centroids);
    hipFree(d_labels);
    hipEventRecord(mem_overhead_stop);
    hipEventSynchronize(mem_overhead_stop);
    hipEventElapsedTime(&temp, mem_overhead_start, mem_overhead_stop);
    mem_overhead_time += temp;
    
    // return times
    float* times = new float[2];
    times[0] = mem_overhead_time;
    times[1] = algo_time;
    return times;
}

// return random floating-point value in [0.0, 1.0)
float rand_float_cuda() {
    return static_cast<float>(rand()) / static_cast<float> ((long long) RAND_MAX+1);
}

// calculates distance of every point to the closest centroid that has already been picked, stores in d_D array
// will be operating once per point
__global__ void kmeansplusplus_kernel(int num_pts, double* d_D, int* d_centroid_indices, double* d_points, int alr_selected, int dims) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index<num_pts) {
        double shortest_dist = DBL_MAX;
        for(int i=0;i<alr_selected;i++) {
            int centroid_index = d_centroid_indices[i];
            double distance = calcDistanceCuda(&d_points[centroid_index*dims], &d_points[index*dims], dims);
            if(distance<shortest_dist) {
                shortest_dist = distance;
            }
        }
        d_D[index] = shortest_dist;
    }
}

int* kmeansplusplus_init_centroids(int num_cluster, double** points, int num_pts, int dims) {
    int* centroid_indices = new int[num_cluster];
    centroid_indices[0] = (int) (rand_float_cuda()*num_pts); // stores indices we have picked so far
    double* D = new double[num_pts]; // stores distances
    int index = 1;

    double* d_points;
    double* d_D;
    int* d_centroid_indices;

    hipMalloc((void**) &d_points, num_pts * dims * sizeof(double));
    hipMalloc((void**) &d_D, num_pts * sizeof(double));
    hipMalloc((void**) &d_centroid_indices, num_cluster * sizeof(int));

    for(int i=0; i<num_pts; i++) {
        hipMemcpy(&d_points[i*dims], points[i], dims * sizeof(double), hipMemcpyHostToDevice);
    }

    while (index < num_cluster) {
        // Here, you`ll need to compute D(x) for all points.
        hipMemcpy(d_centroid_indices, centroid_indices, num_cluster * sizeof(int), hipMemcpyHostToDevice);
        kmeansplusplus_kernel<<<(num_pts+32-1)/32, 32>>>(num_pts, d_D, d_centroid_indices, d_points, index, dims);
        hipDeviceSynchronize();
        hipMemcpy(D, d_D, num_pts * sizeof(double), hipMemcpyDeviceToHost);

        // Choose a new initial centroid
        float total_dist = 0.0;
        for (int i = 0; i < num_pts; i++) {
            total_dist += D[i]*D[i];
        }
        float target = rand_float_cuda() * total_dist;
        float dist = 0.0;
        for (int i = 0; i < num_pts; i++) {
            dist += D[i]*D[i];
            if (target < dist) {
                centroid_indices[index] = i;
                index++;
                break;
            }
        }
    }

    // free all memory
    hipFree(d_points);
    hipFree(d_D);
    hipFree(d_centroid_indices);
    delete[] D;

    return centroid_indices;
}
